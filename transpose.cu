/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

const int TILE_DIM = 32;
const int BLOCK_ROWS = 1;
const int NUM_REPS = 100;

// Check errors and print GB/s
void postprocess(const float *ref, const float *res, int n, float ms)
{
  bool passed = true;
  
  
/*  printf("\nreference:\n");
  for (int i=0; i<2; i++) {
      for (int j=0; j<2; j++) {
          printf("(i,j)=(%d,%d)\n", i,j);
          for (int k=0; k<64; k++)
              printf("%3.0f ", ref[i*2*64 + j * 64 + k]);
          printf("\n");
      }
      printf("\n");
  }
  printf("\n");
  printf("\nresult:\n");
  for (int i=0; i<2; i++) {
      for (int j=0; j<2; j++) {
          printf("(i,j)=(%d,%d)\n",i,j);
          for (int k=0; k<64; k++)
              printf("%3.0f ", res[i*2*64 + j * 64 + k]);
          printf("\n");
      }
      printf("\n");
  }
  printf("\n");
  */
  
  for (int i = 0; i < n; i++)
    if (res[i] != ref[i]) {
      printf("%d %f %f\n", i, res[i], ref[i]);
      printf("%25s\n", "*** FAILED ***");
      passed = false;
      break;
    }
  if (passed)
    printf("%20.2f\n", 2 * n * sizeof(float) * 1e-6 * NUM_REPS / ms );
}

// simple copy kernel
// Used as reference case representing best effective bandwidth.
__global__ void copy(float *odata, const float *idata, int nx, int ny)
{
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  //int width = gridDim.x * TILE_DIM;
  if(!(x<nx && y<ny)) return; 

  for (int j = 0; j < TILE_DIM && y+j < ny; j+= BLOCK_ROWS)
    //odata[(y+j)*width + x] = idata[(y+j)*width + x];
    odata[(y+j)*nx + x] = idata[(y+j)*nx + x];
}

// copy kernel using shared memory
// Also used as reference case, demonstrating effect of using shared memory.
__global__ void copySharedMem(float *odata, const float *idata, int nx, int ny)
{
  __shared__ float tile[TILE_DIM * TILE_DIM];
  
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  //int width = gridDim.x * TILE_DIM;
  if(!(x<nx && y<ny)) return; 


  for (int j = 0; j < TILE_DIM && y+j < ny; j += BLOCK_ROWS)
     tile[(threadIdx.y+j)*TILE_DIM + threadIdx.x] = idata[(y+j)*nx + x];

  __syncthreads();

  for (int j = 0; j < TILE_DIM && y+j < ny; j += BLOCK_ROWS)
     odata[(y+j)*nx + x] = tile[(threadIdx.y+j)*TILE_DIM + threadIdx.x];          
}

// naive transpose for 3D
// Simplest transpose; doesn't use shared memory.
// Global memory reads are coalesced but writes are not.
__global__ void transposeNaive3D(float *odata, const float *idata, const int* sizes, const int* perm, const int dim)
{
  int pos0 = blockIdx.x * TILE_DIM + threadIdx.x;
  int pos1 = blockIdx.y * TILE_DIM + threadIdx.y;
  int pos2 = blockIdx.z * 1 + threadIdx.z;
  const int nx = sizes[0];
  const int ny = sizes[1];
  const int nz = sizes[2];

  if (perm[0] == 1 and perm[1] == 2)   // exchange j, k
      if (pos0<nz && pos1<ny && pos2<nx)
          for (int j = 0; j < TILE_DIM && pos1+j<ny; j += BLOCK_ROWS)
              odata[pos2*nz*ny + pos0*ny + pos1+j] = 
                  idata[pos2*ny*nz + (pos1+j)*nz + pos0];
  if (perm[0] == 0 and perm[1] == 2)   // i, k. pos0:z, pos1:x, pos2:y
      if (pos0<nz && pos1<nx && pos2<ny)
          for (int j = 0; j < TILE_DIM && pos1+j<nx; j += BLOCK_ROWS)
              odata[pos0*nx*ny + pos2*nx + (pos1+j)] = 
                  idata[(pos1+j)*ny*nz + pos2*nz + pos0];
  if (perm[0] == 0 and perm[1] == 1)   // i, j
      if (pos0<nz && pos1<ny && pos2<nx)
          for (int j = 0; j < TILE_DIM && pos1+j<ny; j += BLOCK_ROWS)
              odata[(pos1+j)*nx*nz + pos2*nz + pos0] = 
                  idata[pos2*ny*nz + (pos1+j)*nz + pos0];
}

// coalesced transpose
// Uses shared memory to achieve coalesing in both reads and writes
// Tile width == #banks causes shared memory bank conflicts.

__global__ void transposeCoalesced(float *odata, const float *idata, int nx, int ny)
{
  __shared__ float tile[TILE_DIM][TILE_DIM];
    
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  //int width = gridDim.x * TILE_DIM;
  if(x<nx && y<ny) { 
     for (int j = 0; j < TILE_DIM && y+j < ny; j += BLOCK_ROWS)
        tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*nx + x];
  }
  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;
  if(x<ny && y<nx) {
     for (int j = 0; j < TILE_DIM && y+j < nx; j += BLOCK_ROWS)
        odata[(y+j)*ny + x] = tile[threadIdx.x][threadIdx.y + j];
  }
}
  

// No bank-conflict transpose
// Same as transposeCoalesced except the first tile dimension is padded 
// to avoid shared memory bank conflicts.
__global__ void transposeNoBankConflicts(float *odata, const float *idata, int nx, int ny)
{
  __shared__ float tile[TILE_DIM][TILE_DIM+1];
    
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  //int width = gridDim.x * TILE_DIM;
  if(x<nx && y<ny) { 
     for (int j = 0; j < TILE_DIM && y+j < ny; j += BLOCK_ROWS)
        tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*nx + x];
  }
  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;
  if(x<ny && y<nx) {
     for (int j = 0; j < TILE_DIM && y+j < nx; j += BLOCK_ROWS)
        odata[(y+j)*ny + x] = tile[threadIdx.x][threadIdx.y + j];
  }
}

__global__ void transposeInplace(float *odata, const float *idata, const int* sizes, const int* perm, const int dim)
{
  __shared__ float tile[TILE_DIM][TILE_DIM];

  int idx0 = threadIdx.x;
  int idx1 = threadIdx.y;
  int idx2 = threadIdx.z;
  int pos0 = blockIdx.x * TILE_DIM + idx0;  // transpose block offset
  int pos1 = blockIdx.y * TILE_DIM + idx1;
  int pos2 = blockIdx.z * 1 + idx2;   
  int postpos0 = blockIdx.y * TILE_DIM + idx0;
  int postpos1 = blockIdx.x * TILE_DIM + idx1;
  assert(threadIdx.z==0);
  const int nx = sizes[0];
  const int ny = sizes[1];
  const int nz = sizes[2];

  // idx0 --> z, idx1 --> y, idx2 --> x 
  if (perm[0] == 1 and perm[1] == 2) {  // exchange j, k
      if (pos2<nx && pos0 < nz) {
          for (int j = 0; j < TILE_DIM && pos1+j<ny; j += BLOCK_ROWS)
              tile[idx1+j][(idx0+idx1+j)%TILE_DIM] = 
                  idata[pos2*ny*nz + (pos1+j)*nz + pos0];
      }
      __syncthreads();
      if (pos2<nx && postpos0<ny) {
          for (int j = 0; j < TILE_DIM && postpos1+j<nz; j += BLOCK_ROWS)
              odata[pos2*nz*ny + (postpos1+j)*ny + postpos0] = 
                  tile[idx0][(idx1+j+idx0)%TILE_DIM];
      }
  }

  // idx0: z, idx1: x, idx2: y
  if (perm[0] == 0 and perm[1] == 2) {  // i, k
      if (pos2<ny && pos0<nz) {
          for (int j = 0; j < TILE_DIM && pos1+j<nx; j += BLOCK_ROWS)
              tile[idx1+j][(idx0+idx1+j)%TILE_DIM] = 
                  idata[(pos1+j)*ny*nz + pos2*nz + pos0];
      }
      __syncthreads();
      if (pos2<ny && postpos0 < nx) {
          for (int j = 0; j < TILE_DIM && postpos1+j<nz; j += BLOCK_ROWS)
              odata[(postpos1+j)*ny*nx + pos2*nx + postpos0] = 
                  tile[idx0][(idx1+j+idx0)%TILE_DIM];
      }
  }

  // idx0: z, idx1: y, idx2: x
  if (perm[0] == 0 && perm[1] == 1)  // i, j
      if (pos2<nx && pos0<nz)
          for (int j = 0; j < TILE_DIM && pos1+j<ny; j += BLOCK_ROWS)
              odata[(pos1+j)*nx*nz + pos2*nz + pos0] = 
                  idata[pos2*ny*nz + (pos1+j)*nz + pos0];
}

__host__ __device__ void arridxToNumidx(int& numidx, int* arridx, const int* sizes, const int dim, const int scale) {
   int lower_scale = scale;
   numidx = 0;
   for(int i=0;i<dim;i++){
      lower_scale /= sizes[i];
      numidx += arridx[i]*lower_scale;
   }
}

__host__ __device__ void numidxToArridx(int* arridx, int numidx, const int* sizes, const int dim, const int scale) {
   int lower_scale = scale;
   for(int i=0;i<dim;i++){
      lower_scale /= sizes[i];
      arridx[i] = numidx / lower_scale;
      numidx %= lower_scale;
   }
}

__host__ __device__ int checkRange(const int* arridx, const int* sizes, const int dim) {
   int ret=1;
   for(int i=0;i<dim;i++){
      if(arridx[i] >= sizes[i]) {
         ret=0;
	 break;
      }
   }
   return ret;
}
__global__ void transposeInplaceMultiDim(float *odata, const float *idata, const int* sizes, const int* sizes_perm, const int* perm, const int dim, const int scale, const int magic_scale){
  __shared__ float tile[TILE_DIM][TILE_DIM];
  int idxff = threadIdx.x;
  int idxf = threadIdx.y;
  
  // diagonal reordering
  int bid = blockIdx.x + gridDim.x*blockIdx.y;
  int blockIdx_y = bid%gridDim.y;
  int blockIdx_x = ((bid/gridDim.y)+blockIdx_y)%gridDim.x;

  int posff = blockIdx_x * TILE_DIM + idxff;
  int posf = blockIdx_y * TILE_DIM + idxf;
  int posl = blockIdx.z; 
  int postposff = blockIdx_y * TILE_DIM + idxff;
  int postposf = blockIdx_x * TILE_DIM + idxf;
  int scale_remaining;

  int arridx[10], arridx_out[10], arridx_2[10], sizes_2[10];    // assume maximum dimension=10
  int numidx, numidx_out;
   
  
  if(perm[1]==dim-1) {
      // find the corresponding arridx of this thread
      scale_remaining = scale/sizes[perm[0]]/sizes[perm[1]];
      int d=0;
      for(int j=0;j<dim;j++)
         if(j!=perm[0] && j!=perm[1])
             sizes_2[d++] = sizes[j];
      numidxToArridx(arridx_2, posl, sizes_2, dim-2, scale_remaining);
      d=0;
      for(int j=0;j<dim;j++){
         if(j!=perm[0] && j!=dim-1)
             arridx[j] = arridx_2[d++];
         else if(j==perm[0]) 
	     arridx[j] = posf;
         else
	     arridx[j] = posff;
      } 
      ///////////////////////////////////////////////
      numidx = (posl%magic_scale)*sizes[perm[1]] 
          + (posl/magic_scale)*sizes[perm[0]]*sizes[perm[1]]*magic_scale 
          + posff + posf*sizes[perm[1]]*magic_scale;
      int numidx_incr=BLOCK_ROWS*magic_scale*sizes[perm[1]];
      if (checkRange(arridx, sizes, dim)) {  // can be improved
         for(int j = 0; j < TILE_DIM && posf+j < sizes[perm[0]]; j += BLOCK_ROWS){
            //arridx[perm[0]] = posf+j;   // can be improved !!!!!!!!
            //arridxToNumidx(numidx, arridx, sizes, dim, scale);  // can be improved
	        tile[idxf+j][(idxf+idxff+j)%TILE_DIM] = idata[numidx];
            numidx += numidx_incr;
         } 
      }
      __syncthreads();

      // find the corresponding arridx of this thread
      arridx[dim-1]=postposff;
      arridx[perm[0]]=postposf;
      numidx_out = (posl%magic_scale)*sizes[perm[0]] 
          + (posl/magic_scale)*sizes[perm[0]]*sizes[perm[1]]*magic_scale 
          + postposff + postposf*sizes[perm[0]]*magic_scale;
      ///////////////////////////////////////////////
  
	  int numidx_out_incr = BLOCK_ROWS * magic_scale*sizes[perm[0]];
      if (checkRange(arridx, sizes_perm, dim)) {
         for(int j = 0; j < TILE_DIM && postposf+j < sizes[dim-1]; j += BLOCK_ROWS ){
            //arridx[perm[0]] = postposf+j;
	        //arridxToNumidx(numidx, arridx, sizes_perm, dim, scale);
            odata[numidx_out] = tile[idxff][(idxf+idxff+j)%TILE_DIM];
	        numidx_out += numidx_out_incr;
         }
      }
  }
  else {
      scale_remaining = scale/sizes[dim-1]/sizes[dim-2];
      int d=0;
      for(int j=0;j<dim-2;j++)
          sizes_2[d++] = sizes[j];
      numidxToArridx(arridx_2, posl, sizes_2, dim-2, scale_remaining);
      for(int j=0;j<dim-2;j++){  
	     arridx[j] = arridx_out[j] = arridx_2[j];
      }
      arridx[dim-2] = arridx_out[dim-2] = posf;
      arridx[dim-1] = arridx_out[dim-1] = posff;
      arridx_out[perm[0]] = arridx[perm[1]];
      arridx_out[perm[1]] = arridx[perm[0]];
      numidx = posl*sizes[dim-2]*sizes[dim-1]+posf*sizes[dim-1]+posff;
	  arridxToNumidx(numidx_out, arridx_out, sizes_perm, dim, scale);
      int numidx_incr = BLOCK_ROWS * sizes[dim-1];
      int numidx_out_incr;
      if(perm[1] == dim-2) numidx_out_incr = BLOCK_ROWS * magic_scale*sizes[perm[0]]*sizes[dim-1];
      else numidx_out_incr = BLOCK_ROWS * sizes[dim-1];
      if(checkRange(arridx, sizes, dim)){
          for(int j=0;j<TILE_DIM && posf+j<sizes[dim-2];j+=BLOCK_ROWS){
	         //arridx[dim-2] = posf+j;
	         //if(perm[1] == dim-2) arridx_out[perm[0]] = posf+j;
	         //else arridx_out[dim-2] = posf+j;
	         //arridxToNumidx(numidx, arridx, sizes, dim, scale);
	         //arridxToNumidx(numidx_out, arridx_out, sizes_perm, dim, scale);
	         odata[numidx_out] = idata[numidx];
             numidx += numidx_incr;
             numidx_out += numidx_out_incr;
	      }
      } 
  }
}

// naive transpose
// Simplest transpose; doesn't use shared memory.
// Global memory reads are coalesced but writes are not.

__global__ void transposeNaive(float *odata, const float *idata, const int* sizes, const int* sizes_perm, const int* perm, const int dim, const int scale, const int magic_scale)
{
  int idxff = threadIdx.x;
  int idxf = threadIdx.y;
  
  // diagonal reordering
  int bid = blockIdx.x + gridDim.x*blockIdx.y;
  int blockIdx_y = bid%gridDim.y;
  int blockIdx_x = ((bid/gridDim.y)+blockIdx_y)%gridDim.x;

  int posff = blockIdx_x * TILE_DIM + idxff;
  int posf = blockIdx_y * TILE_DIM + idxf;
  int posl = blockIdx.z; 
  int scale_remaining = scale/sizes[dim-1]/sizes[dim-2];

  int arridx[10], arridx_out[10], arridx_2[10], sizes_2[10];    // assume maximum dimension=10
  int numidx, numidx_out;

  int d=0;
  for(int j=0;j<dim-2;j++)
     sizes_2[d++] = sizes[j];
  numidxToArridx(arridx_2, posl, sizes_2, dim-2, scale_remaining);
  
  for(int j=0;j<dim-2;j++){  
      arridx[j] = arridx_out[j] = arridx_2[j];
  }
  arridx[dim-2] = arridx_out[dim-2] = posf;
  arridx[dim-1] = arridx_out[dim-1] = posff;
  arridx_out[perm[0]] = arridx[perm[1]];
  arridx_out[perm[1]] = arridx[perm[0]];
  numidx = posl*sizes[dim-2]*sizes[dim-1]+posf*sizes[dim-1]+posff;
  arridxToNumidx(numidx_out, arridx_out, sizes_perm, dim, scale);
  int numidx_incr = BLOCK_ROWS * sizes[dim-1];
  int numidx_out_incr;
  if (perm[1] == dim-1 && perm[0]==dim-2)  numidx_out_incr = BLOCK_ROWS;
  else if (perm[1] == dim-1) numidx_out_incr = BLOCK_ROWS * sizes[perm[0]];
  else if(perm[1] == dim-2) numidx_out_incr = BLOCK_ROWS * magic_scale*sizes[perm[0]]*sizes[dim-1];
  else numidx_out_incr = BLOCK_ROWS * sizes[dim-1];
  if(checkRange(arridx, sizes, dim)){
      for(int j=0;j<TILE_DIM && posf+j<sizes[dim-2];j+=BLOCK_ROWS){
          odata[numidx_out] = idata[numidx];
          numidx += numidx_incr;
          numidx_out += numidx_out_incr;
      }
  } 
}

int main(int argc, char **argv)
{
  int dim=0;
  int sizes[dim];
  int perm[2];  // permuted dimensions in ascending order
  int i=1;
  while(i<argc){
     if(strcmp(argv[i],"-d")==0){
        i++;
        dim = atoi(argv[i++]);
     }
     else if(strcmp(argv[i],"-s")==0){
        assert(dim!=0);
        i++;
        for(int j=0;j<dim;j++){
           sizes[j]=atoi(argv[i++]);
        }
     }
     else if(strcmp(argv[i],"-p")==0){
        i++;
        perm[0]=atoi(argv[i++]);
        perm[1]=atoi(argv[i++]);
     }

  }

  if(perm[0] > perm[1]){
     int t=perm[0];
     perm[0]=perm[1];
     perm[1]=t;
  }
  int scale = 1;
  for (int i = 0; i < dim; i++)
      scale *= sizes[i];  
  const int mem_size = scale*sizeof(float);
 
  int magic_scale = 1;
  for (int i=perm[0]+1;i<=perm[1]-1;i++) 
      magic_scale*=sizes[i];

   
  int nff, nf; // faster, fast
  int nff_niv, nf_niv; 
  // should revise when dim > 3
  // exchange (0,1) is different from (0,2) and (1,2) 
  // always assign threadIdx.x to z direction, 
  nff = sizes[dim-1];
  nf = (perm[1]==dim-1)?sizes[perm[0]]:sizes[dim-2]; // if involving last dim, then the other dim; else the one next to the last dim
  nff_niv = sizes[dim-1];
  nf_niv = sizes[dim-2];

  dim3 dimGrid((nff-1)/TILE_DIM+1, (nf-1)/TILE_DIM+1, scale/(nff*nf));
  dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);
  dim3 dimGrid_niv((nff_niv-1)/TILE_DIM+1, (nf_niv-1)/TILE_DIM+1, scale/(nff_niv*nf_niv));
  dim3 dimBlock_niv(TILE_DIM, BLOCK_ROWS, 1);

  int devId = 2;
  // if (argc > 1) devId = atoi(argv[1]);

  hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, devId));
  printf("\nDevice : %s\n", prop.name);
  printf("perm order: (%d, %d)\n", perm[0], perm[1]);
  printf("Matrix size: ");
  for (int i =0; i<dim; i++)
      printf("%d ", sizes[i]);
  printf("\n");
  printf("Block size: %d %d, Tile size: %d %d\n", TILE_DIM, BLOCK_ROWS, TILE_DIM, TILE_DIM);
  printf("dimGrid: %d %d %d. dimBlock: %d %d %d\n",
         dimGrid.x, dimGrid.y, dimGrid.z, dimBlock.x, dimBlock.y, dimBlock.z);
  
  checkCuda( hipSetDevice(devId) );

  float *h_idata = (float*)malloc(mem_size);
  float *h_cdata = (float*)malloc(mem_size);
  float *h_tdata = (float*)malloc(mem_size);
  float *gold    = (float*)malloc(mem_size);
  
  float *d_idata, *d_cdata, *d_tdata;
  checkCuda( hipMalloc(&d_idata, 2*mem_size) );
  checkCuda( hipMemset(d_idata, 0xFF, 2*mem_size) );
  //checkCuda( cudaMalloc(&d_idata, mem_size) );
  checkCuda( hipMalloc(&d_cdata, mem_size) );
  checkCuda( hipMalloc(&d_tdata, mem_size) );
  int *d_sizes, *d_sizes_perm, *d_perm;
  checkCuda( hipMalloc(&d_sizes, dim*sizeof(int)) );
  checkCuda( hipMalloc(&d_sizes_perm, dim*sizeof(int)) );
  checkCuda( hipMalloc(&d_perm, 2*sizeof(int)) );

  // check parameters and calculate execution configuration
  /*
  if (nx % TILE_DIM || ny % TILE_DIM) {
    printf("nx and ny must be a multiple of TILE_DIM\n");
    goto error_exit;
  }

  if (TILE_DIM % BLOCK_ROWS) {
    printf("TILE_DIM must be a multiple of BLOCK_ROWS\n");
    goto error_exit;
  }
  */
    
  // host
  for (int i=0;i<scale;i++) 
      h_idata[i]=i;

  int sizes_perm[dim];
  for (int i=0;i<dim;i++) sizes_perm[i]=sizes[i];
  sizes_perm[perm[0]] = sizes[perm[1]];
  sizes_perm[perm[1]] = sizes[perm[0]];

  int index[dim];
  for (int i=0;i<scale;i++){ 
      numidxToArridx(index, i, sizes_perm, dim, scale);
      int t=index[perm[0]];
      index[perm[0]]=index[perm[1]];
      index[perm[1]]=t;
      int ans_index;
      arridxToNumidx(ans_index, index, sizes, dim, scale);
      gold[i] = h_idata[ans_index];
  }


  // correct result for error checking
  
  // device
  checkCuda( hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpy(d_perm, perm, 2*sizeof(int), hipMemcpyHostToDevice) );
  checkCuda( hipMemcpy(d_sizes, sizes, dim*sizeof(int), hipMemcpyHostToDevice) );
  checkCuda( hipMemcpy(d_sizes_perm, sizes_perm, dim*sizeof(int), hipMemcpyHostToDevice) );
  

  // events for timing
  hipEvent_t startEvent, stopEvent;
  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );
  float ms;

  // ------------
  // time kernels
  // ------------
  printf("%25s%25s\n", "Routine", "Bandwidth (GB/s)");
 /* 
  // ----
  // copy 
  // ----
  printf("%25s", "copy");
  checkCuda( cudaMemset(d_cdata, 0, mem_size) );
  // warm up
  copy<<<dimGrid, dimBlock>>>(d_cdata, d_idata,nx,ny);
  checkCuda( cudaEventRecord(startEvent, 0) );
  for (int i = 0; i < NUM_REPS; i++)
     copy<<<dimGrid, dimBlock>>>(d_cdata, d_idata,nx,ny);
  checkCuda( cudaEventRecord(stopEvent, 0) );
  checkCuda( cudaEventSynchronize(stopEvent) );
  checkCuda( cudaEventElapsedTime(&ms, startEvent, stopEvent) );
  checkCuda( cudaMemcpy(h_cdata, d_cdata, mem_size, cudaMemcpyDeviceToHost) );
  postprocess(h_idata, h_cdata, nx*ny, ms);

  // -------------
  // copySharedMem 
  // -------------
  printf("%25s", "shared memory copy");
  checkCuda( cudaMemset(d_cdata, 0, mem_size) );
  // warm up
  copySharedMem<<<dimGrid, dimBlock>>>(d_cdata, d_idata,nx,ny);
  checkCuda( cudaEventRecord(startEvent, 0) );
  for (int i = 0; i < NUM_REPS; i++)
     copySharedMem<<<dimGrid, dimBlock>>>(d_cdata, d_idata,nx,ny);
  checkCuda( cudaEventRecord(stopEvent, 0) );
  checkCuda( cudaEventSynchronize(stopEvent) );
  checkCuda( cudaEventElapsedTime(&ms, startEvent, stopEvent) );
  checkCuda( cudaMemcpy(h_cdata, d_cdata, mem_size, cudaMemcpyDeviceToHost) );
  postprocess(h_idata, h_cdata, nx * ny, ms);
*/

  // --------------
  // transposeNaive 
  // --------------
  
  printf("%25s", "naive transpose");
  checkCuda( hipMemset(d_tdata, 0, mem_size) );
  // warmup
  transposeNaive<<<dimGrid_niv, dimBlock_niv>>>(d_tdata, d_idata, d_sizes, d_sizes_perm, d_perm, dim, scale, magic_scale);
  checkCuda( hipEventRecord(startEvent, 0) );
  for (int i = 0; i < NUM_REPS; i++)
     transposeNaive<<<dimGrid_niv, dimBlock_niv>>>(d_tdata, d_idata, d_sizes, d_sizes_perm, d_perm, dim, scale, magic_scale);
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  checkCuda( hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost) );
  postprocess(gold, h_tdata, scale, ms);
/*
  // ------------------
  // transposeCoalesced 
  // ------------------
  printf("%25s", "coalesced transpose");
  checkCuda( cudaMemset(d_tdata, 0, mem_size) );
  // warmup
  transposeCoalesced<<<dimGrid, dimBlock>>>(d_tdata, d_idata,nx,ny);
  checkCuda( cudaEventRecord(startEvent, 0) );
  for (int i = 0; i < NUM_REPS; i++)
     transposeCoalesced<<<dimGrid, dimBlock>>>(d_tdata, d_idata,nx,ny);
  checkCuda( cudaEventRecord(stopEvent, 0) );
  checkCuda( cudaEventSynchronize(stopEvent) );
  checkCuda( cudaEventElapsedTime(&ms, startEvent, stopEvent) );
  checkCuda( cudaMemcpy(h_tdata, d_tdata, mem_size, cudaMemcpyDeviceToHost) );
  postprocess(gold, h_tdata, nx * ny, ms);

  // ------------------------
  // transposeNoBankConflicts
  // ------------------------
  printf("%25s", "conflict-free transpose");
  checkCuda( cudaMemset(d_tdata, 0, mem_size) );
  // warmup
  transposeNoBankConflicts<<<dimGrid, dimBlock>>>(d_tdata, d_idata,nx,ny);
  checkCuda( cudaEventRecord(startEvent, 0) );
  for (int i = 0; i < NUM_REPS; i++)
     transposeNoBankConflicts<<<dimGrid, dimBlock>>>(d_tdata, d_idata,nx,ny);
  checkCuda( cudaEventRecord(stopEvent, 0) );
  checkCuda( cudaEventSynchronize(stopEvent) );
  checkCuda( cudaEventElapsedTime(&ms, startEvent, stopEvent) );
  checkCuda( cudaMemcpy(h_tdata, d_tdata, mem_size, cudaMemcpyDeviceToHost) );
  postprocess(gold, h_tdata, nx * ny, ms);
*/
  // ------------------------
  // transposeInplace
  // ------------------------
  /*
  printf("%25s", "In-place transpose");
  checkCuda( cudaMemset(d_tdata, 0, mem_size) );
  // warmup
  transposeInplace<<<dimGrid, dimBlock>>>(d_tdata, d_idata, d_sizes, d_perm, dim);
  checkCuda( cudaEventRecord(startEvent, 0) );
  for (int i = 0; i < NUM_REPS; i++)
     transposeInplace<<<dimGrid, dimBlock>>>(d_tdata, d_idata, d_sizes, d_perm, dim);
  checkCuda( cudaEventRecord(stopEvent, 0) );
  checkCuda( cudaEventSynchronize(stopEvent) );
  checkCuda( cudaEventElapsedTime(&ms, startEvent, stopEvent) );
  checkCuda( cudaMemcpy(h_tdata, d_tdata, mem_size, cudaMemcpyDeviceToHost) );
  postprocess(gold, h_tdata, scale, ms);
  */
  // ------------------------
  // transposeInplaceMultiDimension
  // ------------------------
  
     printf("%25s", "In-place last dim");
     checkCuda( hipMemset(d_tdata, 0, mem_size) );
     // warmup
     transposeInplaceMultiDim<<<dimGrid, dimBlock>>>(d_tdata, d_idata, d_sizes, d_sizes_perm, d_perm, dim, scale, magic_scale);
     checkCuda( hipEventRecord(startEvent, 0) );
     for (int i = 0; i < NUM_REPS; i++)
        transposeInplaceMultiDim<<<dimGrid, dimBlock>>>(d_tdata, d_idata, d_sizes, d_sizes_perm, d_perm, dim, scale, magic_scale);
     checkCuda( hipEventRecord(stopEvent, 0) );
     checkCuda( hipEventSynchronize(stopEvent) );
     checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
     checkCuda( hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost) );
     postprocess(gold, h_tdata, scale, ms);
  
  
  // cleanup
  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
  checkCuda( hipFree(d_tdata) );
  checkCuda( hipFree(d_cdata) );
  checkCuda( hipFree(d_idata) );
  free(h_idata);
  free(h_tdata);
  free(h_cdata);
  free(gold);
}
