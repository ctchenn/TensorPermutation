/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;
const int NUM_REPS = 100;

// Check errors and print GB/s
void postprocess(const float *ref, const float *res, int n, float ms)
{
  bool passed = true;
  for (int i = 0; i < n; i++)
    if (res[i] != ref[i]) {
      printf("%d %f %f\n", i, res[i], ref[i]);
      printf("%25s\n", "*** FAILED ***");
      passed = false;
      break;
    }
  if (passed)
    printf("%20.2f\n", 2 * n * sizeof(float) * 1e-6 * NUM_REPS / ms );
}

// simple copy kernel
// Used as reference case representing best effective bandwidth.
__global__ void copy(float *odata, const float *idata)
{
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
    odata[(y+j)*width + x] = idata[(y+j)*width + x];
}

// copy kernel using shared memory
// Also used as reference case, demonstrating effect of using shared memory.
__global__ void copySharedMem(float *odata, const float *idata)
{
  __shared__ float tile[TILE_DIM * TILE_DIM];
  
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[(threadIdx.y+j)*TILE_DIM + threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     odata[(y+j)*width + x] = tile[(threadIdx.y+j)*TILE_DIM + threadIdx.x];          
}

// naive transpose
// Simplest transpose; doesn't use shared memory.
// Global memory reads are coalesced but writes are not.
__global__ void transposeNaive(float *odata, const float *idata)
{
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
    odata[x*width + (y+j)] = idata[(y+j)*width + x];
}

// coalesced transpose
// Uses shared memory to achieve coalesing in both reads and writes
// Tile width == #banks causes shared memory bank conflicts.

__global__ void transposeCoalesced(float *odata, const float *idata)
{
  __shared__ float tile[TILE_DIM][TILE_DIM];
    
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}
  

// No bank-conflict transpose
// Same as transposeCoalesced except the first tile dimension is padded 
// to avoid shared memory bank conflicts.
__global__ void transposeNoBankConflicts(float *odata, const float *idata)
{
  __shared__ float tile[TILE_DIM][TILE_DIM+1];
    
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}

__global__ void transposeInplace(float *odata, const float *idata, const int* sizes, const int* perm, const int dim)
{
  __shared__ float tile[TILE_DIM][TILE_DIM];

  int idx1 = threadIdx.x;
  int idx2 = threadIdx.y;
  int idx3 = threadIdx.z;
  int pos1 = blockIdx.x * TILE_DIM + idx1;  // transpose block offset
  int pos2 = blockIdx.y * TILE_DIM + idx2;
  int pos3 = blockIdx.z * 1 + idx3;   
  int postpos1 = blockIdx.y * TILE_DIM + idx1;
  int postpos2 = blockIdx.x * TILE_DIM + idx2;
  assert(threadIdx.z==0);
  const int nx = sizes[0];
  const int ny = sizes[1];
  const int nz = sizes[2];

  // idx1 --> z, idx2 --> y, idx3 --> x 
  if (perm[0] == 1 and perm[1] == 2) {  // exchange j, k
      for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
          tile[idx2+j][(idx1+idx2+j)%TILE_DIM] = 
              idata[pos3*ny*nz + (pos2+j)*nz + pos1];
      __syncthreads();
      for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
          odata[pos3*nz*ny + (postpos2+j)*ny + postpos1] = 
              tile[idx1][(idx2+j+idx1)%TILE_DIM];
  }

  // idx1: z, idx2: x, idx3: y
  if (perm[0] == 0 and perm[1] == 2) {  // i, k
      for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
          tile[idx2+j][(idx1+idx2+j)%TILE_DIM] = 
              idata[(pos2+j)*ny*nz + pos3*nz + pos1];
      __syncthreads();
      for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
          odata[(postpos2+j)*ny*nx + pos3*nx + postpos1] = 
              tile[idx1][(idx2+j+idx1)%TILE_DIM];
  }

  // idx1: y, idx2: x, idx3: z
  if (perm[0] == 0 && perm[1] == 1) { // i, j
      for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
          odata[pos1*nx*nz + (pos2+j)*nz + pos3] = 
              idata[(pos2+j)*ny*nz + pos1*nz + pos3];
  }

}

int main(int argc, char **argv)
{
  const int dim = 3;
  const int sizes[3] = {64,64,64};
  const int perm[2] = {0, 1};  // permuted dimensions in ascending order
  int scale = 1;
  for (int i = 0; i < dim; i++)
      scale *= sizes[i];  
  const int mem_size = scale*sizeof(float);
  
  int nx = sizes[0];
  int ny = sizes[1];
  int nz = sizes[2];

  // should revise when dim > 3
  int n1=sizes[perm[1]];
  int n2=sizes[perm[0]];
  int n3=sizes[0]+sizes[1]+sizes[2]-n1-n2; 

  dim3 dimGrid(n1/TILE_DIM, n2/TILE_DIM, n3);
  dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);

  int devId = 0;
  if (argc > 1) devId = atoi(argv[1]);

  hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, devId));
  printf("\nDevice : %s\n", prop.name);
  printf("Matrix size: %d %d %d, Block size: %d %d, Tile size: %d %d\n", 
         nx, ny, nz, TILE_DIM, BLOCK_ROWS, TILE_DIM, TILE_DIM);
  printf("dimGrid: %d %d %d. dimBlock: %d %d %d\n",
         dimGrid.x, dimGrid.y, dimGrid.z, dimBlock.x, dimBlock.y, dimBlock.z);
  
  checkCuda( hipSetDevice(devId) );

  float *h_idata = (float*)malloc(mem_size);
  float *h_cdata = (float*)malloc(mem_size);
  float *h_tdata = (float*)malloc(mem_size);
  float *gold    = (float*)malloc(mem_size);
  
  float *d_idata, *d_cdata, *d_tdata;
  checkCuda( hipMalloc(&d_idata, mem_size) );
  checkCuda( hipMalloc(&d_cdata, mem_size) );
  checkCuda( hipMalloc(&d_tdata, mem_size) );
  int *d_sizes, *d_perm;
  checkCuda( hipMalloc(&d_sizes, 3*sizeof(int)) );
  checkCuda( hipMalloc(&d_perm, 2*sizeof(int)) );

  // check parameters and calculate execution configuration
  if (nx % TILE_DIM || ny % TILE_DIM) {
    printf("nx and ny must be a multiple of TILE_DIM\n");
    goto error_exit;
  }

  if (TILE_DIM % BLOCK_ROWS) {
    printf("TILE_DIM must be a multiple of BLOCK_ROWS\n");
    goto error_exit;
  }
    
  // host
  for (int i = 0; i < nx; i++)
      for (int j = 0; j < ny; j++)
          for (int k = 0; k < nz; k++)
              h_idata[i*ny*nz + j*nz + k] = i*ny*nz + j*nz + k;

  // correct result for error checking
  for (int i = 0; i < nx; i++)
      for (int j = 0; j < ny; j++)
          for (int k = 0; k < nz; k++) {
              if (perm[0] == 1 && perm[1] == 2)  // exchange j, k
                  gold[i*ny*nz + k*ny + j] = h_idata[i*ny*nz + j*nz + k];
              else if(perm[0] == 0 && perm[1] == 2)  // i, k
                  gold[k*nx*ny + j*nx + i] = h_idata[i*ny*nz + j*nz + k];
              else if(perm[0] == 0 && perm[1] == 1)  // i, j
                  gold[j*nz*nx + i*nz + k] = h_idata[i*ny*nz + j*nz + k];
          } 
  
  // device
  checkCuda( hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpy(d_perm, perm, 2*sizeof(int), hipMemcpyHostToDevice) );
  checkCuda( hipMemcpy(d_sizes, sizes, 3*sizeof(int), hipMemcpyHostToDevice) );
  
  // events for timing
  hipEvent_t startEvent, stopEvent;
  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );
  float ms;

  // ------------
  // time kernels
  // ------------
  printf("%25s%25s\n", "Routine", "Bandwidth (GB/s)");
 /* 
  // ----
  // copy 
  // ----
  printf("%25s", "copy");
  checkCuda( cudaMemset(d_cdata, 0, mem_size) );
  // warm up
  copy<<<dimGrid, dimBlock>>>(d_cdata, d_idata);
  checkCuda( cudaEventRecord(startEvent, 0) );
  for (int i = 0; i < NUM_REPS; i++)
     copy<<<dimGrid, dimBlock>>>(d_cdata, d_idata);
  checkCuda( cudaEventRecord(stopEvent, 0) );
  checkCuda( cudaEventSynchronize(stopEvent) );
  checkCuda( cudaEventElapsedTime(&ms, startEvent, stopEvent) );
  checkCuda( cudaMemcpy(h_cdata, d_cdata, mem_size, cudaMemcpyDeviceToHost) );
  postprocess(h_idata, h_cdata, nx*ny, ms);

  // -------------
  // copySharedMem 
  // -------------
  printf("%25s", "shared memory copy");
  checkCuda( cudaMemset(d_cdata, 0, mem_size) );
  // warm up
  copySharedMem<<<dimGrid, dimBlock>>>(d_cdata, d_idata);
  checkCuda( cudaEventRecord(startEvent, 0) );
  for (int i = 0; i < NUM_REPS; i++)
     copySharedMem<<<dimGrid, dimBlock>>>(d_cdata, d_idata);
  checkCuda( cudaEventRecord(stopEvent, 0) );
  checkCuda( cudaEventSynchronize(stopEvent) );
  checkCuda( cudaEventElapsedTime(&ms, startEvent, stopEvent) );
  checkCuda( cudaMemcpy(h_cdata, d_cdata, mem_size, cudaMemcpyDeviceToHost) );
  postprocess(h_idata, h_cdata, nx * ny, ms);

  // --------------
  // transposeNaive 
  // --------------
  printf("%25s", "naive transpose");
  checkCuda( cudaMemset(d_tdata, 0, mem_size) );
  // warmup
  transposeNaive<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
  checkCuda( cudaEventRecord(startEvent, 0) );
  for (int i = 0; i < NUM_REPS; i++)
     transposeNaive<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
  checkCuda( cudaEventRecord(stopEvent, 0) );
  checkCuda( cudaEventSynchronize(stopEvent) );
  checkCuda( cudaEventElapsedTime(&ms, startEvent, stopEvent) );
  checkCuda( cudaMemcpy(h_tdata, d_tdata, mem_size, cudaMemcpyDeviceToHost) );
  postprocess(gold, h_tdata, nx * ny, ms);

  // ------------------
  // transposeCoalesced 
  // ------------------
  printf("%25s", "coalesced transpose");
  checkCuda( cudaMemset(d_tdata, 0, mem_size) );
  // warmup
  transposeCoalesced<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
  checkCuda( cudaEventRecord(startEvent, 0) );
  for (int i = 0; i < NUM_REPS; i++)
     transposeCoalesced<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
  checkCuda( cudaEventRecord(stopEvent, 0) );
  checkCuda( cudaEventSynchronize(stopEvent) );
  checkCuda( cudaEventElapsedTime(&ms, startEvent, stopEvent) );
  checkCuda( cudaMemcpy(h_tdata, d_tdata, mem_size, cudaMemcpyDeviceToHost) );
  postprocess(gold, h_tdata, nx * ny, ms);

  // ------------------------
  // transposeNoBankConflicts
  // ------------------------
  printf("%25s", "conflict-free transpose");
  checkCuda( cudaMemset(d_tdata, 0, mem_size) );
  // warmup
  transposeNoBankConflicts<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
  checkCuda( cudaEventRecord(startEvent, 0) );
  for (int i = 0; i < NUM_REPS; i++)
     transposeNoBankConflicts<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
  checkCuda( cudaEventRecord(stopEvent, 0) );
  checkCuda( cudaEventSynchronize(stopEvent) );
  checkCuda( cudaEventElapsedTime(&ms, startEvent, stopEvent) );
  checkCuda( cudaMemcpy(h_tdata, d_tdata, mem_size, cudaMemcpyDeviceToHost) );
  postprocess(gold, h_tdata, nx * ny, ms);
*/
  // ------------------------
  // transposeInplace
  // ------------------------
  printf("%25s", "In-place transpose");
  checkCuda( hipMemset(d_tdata, 0, mem_size) );
  // warmup
  transposeInplace<<<dimGrid, dimBlock>>>(d_tdata, d_idata, d_sizes, d_perm, dim);
  checkCuda( hipEventRecord(startEvent, 0) );
  for (int i = 0; i < NUM_REPS; i++)
     transposeInplace<<<dimGrid, dimBlock>>>(d_tdata, d_idata, d_sizes, d_perm, dim);
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  checkCuda( hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost) );
  postprocess(gold, h_tdata, nx*ny*nz, ms);
error_exit:
  // cleanup
  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
  checkCuda( hipFree(d_tdata) );
  checkCuda( hipFree(d_cdata) );
  checkCuda( hipFree(d_idata) );
  free(h_idata);
  free(h_tdata);
  free(h_cdata);
  free(gold);
}
